
#include <hip/hip_runtime.h>
// Copyright 2010 The MathWorks, Inc.
// $Revision: 1.1.8.2 $   $Date: 2010/05/10 17:03:57 $

/*
 * Write only bandwith test - compute the index that this thread will read
 * and if the second input is greater than zero write it to the output
 */
__global__ void bandwidth1(float * pOutput, float val ) {
    // Calculate (for each thread) which element of the array to write
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if ( val > 0 ) {
        pOutput[idx] = val;
    }
}

/*
 * Read and write only bandwith test - compute the index that this thread 
 * will read and if the second input is greater than zero read from the 
 * input and add 
 */
__global__ void bandwidth2(float * pData, float val ) {
    // Calculate (for each thread) which element of the array to read/write
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if ( val > 0 ) {
        pData[idx] = pData[idx] + val;
    }
}

