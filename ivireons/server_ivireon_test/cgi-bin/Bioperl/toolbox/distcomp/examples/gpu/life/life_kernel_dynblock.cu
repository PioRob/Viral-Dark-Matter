
#include <hip/hip_runtime.h>
// Copyright 2010 The MathWorks, Inc.
// $Revision: 1.1.8.1 $   $Date: 2010/05/03 16:03:48 $

// A simple function to map co-ordinates to linear indices.
__device__ unsigned int iLinOffSet( int r, int c, 
                                    int m, int n,
                                    int offRow, int offCol ) {
    r = r + offRow;
    r = ( r >= m ? r - m : r );
    r = ( r <  0 ? r + m : r );
 
    c = c + offCol;
    c = ( c >= n ? c - n : c );
    c = ( c <  0 ? c + n : c );
    return c * m + r;
}

// Shared memory used to communicate between threads for stencil calculation
extern __shared__ unsigned char block[];

// macro to simplify accessing shared memory
#define BLOCK_EL( yyy, xxx ) block[ iLinOffSet( (yyy), (xxx), blockDim.y, blockDim.x, 0, 0 ) ]

// one generation of the game of life. Overwrites board each turn.
__global__ void life( unsigned char * board, int m, int n ) {

    // co-ords in global board
    int ix     = blockIdx.x * blockDim.x + threadIdx.x;
    int iy     = blockIdx.y * blockDim.y + threadIdx.y;
    int linidx = iLinOffSet( iy, ix, m, n, 0, 0 );

    // Load up the shared memory - plus ghost cells using toroidal boundary
    // conditions.
    BLOCK_EL( threadIdx.y + 1, threadIdx.x + 1 ) = board[linidx];
    if ( threadIdx.x == 0 ) {
        BLOCK_EL( threadIdx.y, 0 ) = board[ iLinOffSet( iy, ix, m, n, 0, -1 ) ];
    }
    if ( threadIdx.x == blockDim.x - 1 ) {
        BLOCK_EL( threadIdx.y, blockDim.x + 1 ) = board[ iLinOffSet( iy, ix, m, n, 0, 1 ) ];
    }
    if ( threadIdx.y == 0 ) {
        BLOCK_EL( 0, threadIdx.x ) = board[ iLinOffSet( iy, ix, m, n, -1, 0 ) ];
    }
    if ( threadIdx.y == blockDim.y - 1 ) {
        BLOCK_EL( blockDim.y + 1, threadIdx.x ) = board[ iLinOffSet( iy, ix, m, n, 1, 0 ) ];
    }

    // Make sure all shared memory is loaded
    __syncthreads();
    
    // Game of life stencil computation
    int liveNeighbours = 0;
    int imAlive        = BLOCK_EL( threadIdx.y + 1, threadIdx.x + 1 );
    for ( int xoff = 0; xoff <= 2; xoff++ ) {
        for ( int yoff = 0; yoff <= 2; yoff++ ) {
            if ( ! ( xoff == 1 && yoff == 1 ) ) {
                liveNeighbours += BLOCK_EL( threadIdx.y + yoff, threadIdx.x + xoff );
            }
        }
    }
    
    // Finally, update the board.
    if ( ix < n && iy < m ) {
        board[linidx] = ( imAlive && liveNeighbours == 2 ||
                          liveNeighbours == 3 );
    }
}
