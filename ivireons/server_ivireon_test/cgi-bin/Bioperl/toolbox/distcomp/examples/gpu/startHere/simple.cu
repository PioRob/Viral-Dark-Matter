
#include <hip/hip_runtime.h>
/*
 * To compile this CU file use
 *
 *  nvcc -ptx simple.cu
 *
 * This will generate simple.ptx in the same directory. Then the kernels
 * can be loaded with makeKernel using
 *
 * k = parallel.gpu.CUDAKernel('simple.ptx', 'simple.cu', entryName);
 *
 * Make sure you are in the same dir as this file.
 */

// Copyright 2010 The MathWorks, Inc.
// $Revision: 1.1.8.2 $   $Date: 2010/05/10 17:04:02 $

/*
 * Define a very simple kernel to run on a single thread that adds a float
 * to another one. NOTE that any outputs MUST be pointers, hence the first
 * input to this function is a pointer, so that MATLAB treats it as an 
 * output.
 */
__global__ void reallySimple( float * pi, float c ) {
    *pi += c;
}

/*
 * Lets now use a thread block to run many threads at once. We will given
 * a vector as input and assume that there are the correct number of threads
 * for array elements. Each thread will add the constant c to each element
 */
__global__ void usesThreadBlock( float * pi, float c )  {
    int idx = threadIdx.x;
    pi[idx] += c;
}

/*
 * Lets now use both a thread block and a grid to go bigger than 512 in size
 */
__global__ void usesGridsAndBlocks( float * pi, float c )  {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    pi[idx] += c;
}

/*
 * Example that works correctly when there are more threads than array
 * elements.
 */
__global__ void includeArraySize( float * pi, float c, int s )  {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if ( idx < s ) {
        pi[idx] += c;
    }
}
